#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <stdlib.h>

#include <newsap/common.h>

#ifdef USE_OLD_CUSP
#  include <cusp/blas.h>
#else
#  include <cusp/blas/blas.h>
#endif
#include <cusp/array1d.h>
#include <cusp/print.h>

#include <newsap/segmented_matrix.h>

typedef double REAL;
typedef typename cusp::array1d<REAL, cusp::device_memory>             Vector;
typedef typename cusp::array1d<REAL, cusp::host_memory>               VectorH;
typedef typename cusp::array1d<int, cusp::device_memory>              IntVector;
typedef typename cusp::array1d<int, cusp::host_memory>                IntVectorH;
typedef typename newsap::SegmentedMatrix<REAL, cusp::device_memory>   Matrix;

using std::cout;
using std::cerr;
using std::cin;
using std::endl;

// -----------------------------------------------------------------------------
// Macro to obtain a random number between two specified values
// -----------------------------------------------------------------------------
#define RAND(L, H) ((L) + ((H) - (L)) * (float)rand() / (float)RAND_MAX)


// -----------------------------------------------------------------------------
// Definitions for SimpleOpt and SimpleGlob
// -----------------------------------------------------------------------------
#include <SimpleOpt/SimpleOpt.h>

// ID values to identify command line arguments
enum {OPT_HELP, OPT_BAND};

// Table of CSimpleOpt::Soption structures. Each entry specifies:
// - the ID for the option (returned from OptionId() during processing)
// - the option as it should appear on the command line
// - type of the option
// The last entry must be SO_END_OF_OPTIONS
CSimpleOptA::SOption g_options[] = {
	{ OPT_BAND,          "-b",                   SO_MULTI   },
	{ OPT_BAND,          "--banded-synthetic",   SO_MULTI   },
	{ OPT_HELP,          "-?",                   SO_NONE    },
	{ OPT_HELP,          "-h",                   SO_NONE    },
	{ OPT_HELP,          "--help",               SO_NONE    },
	SO_END_OF_OPTIONS
};

void ShowUsage();
bool
GetProblemSpecs(int             argc, 
                char**          argv,
                int&            N,
                int&            k,
                REAL&           d);

void
GetSegmentedMatrices(
    int N,
    int k,
    REAL d,
    IntVectorH& num_rows,
    IntVectorH& num_columns,
    IntVectorH& A_offsets,
    IntVectorH& B_offsets,
    IntVectorH& C_offsets,
    VectorH&    A,
    VectorH&    B,
    VectorH&    C
);

int main(int argc, char **argv) {
    int            pN;
    int            pk;
    REAL           pd;

	if (!GetProblemSpecs(argc, argv, pN, pk, pd)) {
        ShowUsage();
        return 1;
    }

    unsigned num_partitions = pN / pk;

    IntVectorH       num_rows, num_columns;
    IntVectorH       A_offsets, B_offsets, C_offsets;
    VectorH          subA, subB, subC;

    GetSegmentedMatrices(
        pN,
        pk,
        pd,
        num_rows,
        num_columns,
        A_offsets,
        B_offsets,
        C_offsets,
        subA,
        subB,
        subC
    );
    cusp::print(A_offsets);
    cusp::print(subA);
    cusp::print(B_offsets);
    cusp::print(subB);
    cusp::print(C_offsets);
    cusp::print(subC);

    Matrix A(num_partitions, num_rows, num_columns, A_offsets, B_offsets, C_offsets, subA, subB, subC);

    return 0;
}

void ShowUsage()
{
	cout << "Usage:  driver_mm [OPTIONS]" << endl;
	cout << endl;
	cout << " -b SIZE BW DD" << endl;
	cout << " --banded-synthetic SIZE BW DD" << endl;
	cout << "        Use a synthetic banded matrix of size SIZE, half-bandwidth BW," << endl;
	cout << "        and degree of diagonal dominance DD." << endl;
}

// -----------------------------------------------------------------------------
// GetProblemSpecs()
//
// This function parses the specified program arguments and sets up the problem
// to be solved.
// -----------------------------------------------------------------------------
bool
GetProblemSpecs(int             argc, 
                char**          argv,
                int&            N,
                int&            k,
                REAL&           d)
{
    N = -1;
    k = -1;
    d = -1.0;
	// Create the option parser and pass it the program arguments and the array
	// of valid options. Then loop for as long as there are arguments to be
	// processed.
	CSimpleOptA args(argc, argv, g_options);

	while (args.Next()) {
		// Exit immediately if we encounter an invalid argument.
		if (args.LastError() != SO_SUCCESS) {
			cout << "Invalid argument: " << args.OptionText() << endl;
			ShowUsage();
			return false;
		}

		// Process the current argument.
		switch (args.OptionId()) {
			case OPT_HELP:
				return false;
			case OPT_BAND:
				{
					char **mArgs = args.MultiArg(3);
					if (!mArgs) {
						return false;
					}
					N = atoi(mArgs[0]);
					k = atoi(mArgs[1]);
					d = atof(mArgs[2]);

					break;
				}
		}
	}

    if (N < 0 || k < 0 || d < 0) {
        ShowUsage();
        return false;
    }

	return true;
}

void
GetSegmentedMatrices(
    int N,
    int k,
    REAL d,
    IntVectorH& num_rows,
    IntVectorH& num_columns,
    IntVectorH& A_offsets,
    IntVectorH& B_offsets,
    IntVectorH& C_offsets,
    VectorH&    A,
    VectorH&    B,
    VectorH&    C
) {
    int num_partitions = N / k;
    int remainder = N % num_partitions;

    num_rows.resize(num_partitions);
    num_columns.resize(num_partitions);
    A_offsets.resize(num_partitions + 1);
    B_offsets.resize(num_partitions + 1);
    C_offsets.resize(num_partitions + 1);

    int cur_a_offset = 0;
    int cur_b_offset = 0;
    int cur_c_offset = 0;

    for (int i = 0; i < num_partitions; i++) {
        A_offsets[i] = cur_a_offset;
        B_offsets[i] = cur_b_offset;
        C_offsets[i] = cur_c_offset;
        if (i < remainder) {
            num_rows[i] = num_columns[i] = k + 1;
            cur_a_offset += (k + 1) * (k + 1);
            cur_b_offset += ((i > 0) ? ((k + 1) * (k + 1)) : 0);
            cur_c_offset += (k + 1) * ((i < remainder - 1) ? (k + 1) : k);
        } else {
            num_rows[i] = num_columns[i] = k;
            cur_a_offset += k * k;
            cur_b_offset += k * ((i > remainder) ? k : (i == 0 ? 0 : (k + 1)));
            cur_c_offset += k * ((i == (num_partitions - 1)) ? 0 : k);
        }
    }

    A_offsets[num_partitions] = cur_a_offset;
    B_offsets[num_partitions] = cur_b_offset;
    C_offsets[num_partitions] = cur_c_offset;

    A.resize(cur_a_offset, REAL(0));
    B.resize(cur_b_offset, REAL(0));
    C.resize(cur_c_offset, REAL(0));

    for (int i = 0; i < num_partitions; i++) {
        int start_row, end_row;
        int a_num_columns, b_num_columns, c_num_columns;
        if (i < remainder) {
            start_row = (k + 1) * i;
            end_row = start_row + (k + 1);
            a_num_columns = k + 1;
            b_num_columns = (i > 0 ? (k + 1) : 0);
            c_num_columns = (i < remainder - 1 ? (k + 1) : k);
        } else {
            start_row = k * i + remainder;
            end_row = start_row + k;
            a_num_columns = k;
            b_num_columns = (i > remainder ? k : (i == 0 ? 0 : (k + 1)));
            c_num_columns = (i < (num_partitions - 1) ? k : 0);
        }

        REAL my_sum = REAL(0);
        for (int j = start_row; j < end_row; j++) {
            int column_min = j - k;
            int column_max = j + k;
            if (column_min < 0) {
                column_min = 0;
            }
            if (column_max >= N) {
                column_max = N - 1;
            }

            for (int l = column_min; l <= column_max; l++) {
                REAL value = RAND(-10.0, 10.0);
                my_sum += (value < 0 ? -value : value);

                if (l < start_row) {
                    B[B_offsets[i] + (j - start_row + 1) * b_num_columns + l - start_row] = value;
                } else if (l >= end_row) {
                    C[C_offsets[i] + (j - start_row) * c_num_columns + l - end_row] = value;
                } else {
                    A[A_offsets[i] + (j - start_row) * a_num_columns + l - start_row] = value;
                }
            }
            A[A_offsets[i] + (j - start_row) * a_num_columns + j - start_row] = my_sum * d;
        }
    }
}
